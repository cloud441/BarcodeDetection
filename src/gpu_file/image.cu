#include "hip/hip_runtime.h"
#include "image.hh"

#include <iostream>

__global__ void compute_gray(unsigned char *d_gray_array,
            unsigned char *d_array, int img_width, int img_weight,
            int blockSize, int gridSize)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int img_size = img_width * img_weight;

    while (id < img_size)
    {
        d_gray_array[id] = d_array[id * 3] * 0.2989 +
            d_array[id * 3 + 1] * 0.5870 + d_array[id * 3 + 2] * 0.1140;

        id += blockSize * gridSize;
    }

}



Image::Image(const char* path)
{

    img_array = stbi_load(path, &width, &height, &nb_chan, 0);

    if (img_array == NULL)
    {
        std::cout << "Error : can't open the image: " << path << "\n";
    }

    img_gray_array = new unsigned char[width * height];
}

Image::~Image()
{
    if (img_array)
    {
        stbi_image_free(img_array);
    }

    free(img_gray_array);
}

void Image::save_gray_img()
{
    stbi_write_jpg("../../img/codebar_gray_result.jpg", width, height, 1, img_gray_array, 100);
}

void Image::create_gray_array()
{

    // CPU Version
    for (int i = 0; i < height * width; ++i)
    {
        /*
        img_gray_array[i] = img_array[i * 3] * 0.2989 +
            img_array[i * 3 + 1] * 0.5870 + img_array[i * 3 + 2] * 0.1140;
        */
        img_gray_array[i] = (uint8_t)((img_array[i * 3] + img_array[i * 3 + 1] + img_array[i * 3 + 2])/3.0);
    }

}

int Image::get_size()
{
    return width * height;
}


int main(void)
{
    //Image image("../../img/codebar.jpg");
    //image.create_gray_array();
    //image.save_gray_img();


    int width, height, nb_chan;
    unsigned char *img = stbi_load("../../img/codebar.jpg", &width, &height, &nb_chan, 0);

    size_t img_size = width * height * nb_chan;
    printf("We have a size of %dx%d, %d\n", width, height, img_size);
    int gray_chan = 1;
    size_t gray_img_size = width * height * gray_chan;

    unsigned char *gray_img = (unsigned char *) malloc(gray_img_size);

    for (unsigned char *p = img, *pg = gray_img; p != (img + img_size); p += nb_chan, pg += gray_chan)
    {
        *pg = (uint8_t)((*p + *(p + 1) + *(p + 2))/3.0);
    }

    stbi_write_jpg("../../img/img_gray_result.jpg", width, height, gray_chan, gray_img, 100);

    return 0;
}
