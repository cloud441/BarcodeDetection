#include "hip/hip_runtime.h"
#include "image.hh"

#include <iostream>

__global__ void compute_gray(unsigned char *d_gray_array,
            unsigned char *d_array, int img_width, int img_weight,
            int blockSize, int gridSize)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int img_size = img_width * img_weight;

    while (id < img_size)
    {
        d_gray_array[id] = d_array[id * 3] * 0.2989 +
            d_array[id * 3 + 1] * 0.5870 + d_array[id * 3 + 2] * 0.1140;

        id += blockSize * gridSize;
    }

}



Image::Image(const char* path)
{

    img_array = stbi_load(path, &width, &height, &nb_chan, 0);

    if (img_array == NULL)
    {
        std::cout << "Error : can't open the image: " << path << "\n";
    }

    img_gray_array = new unsigned char[width * height];
}

Image::~Image()
{
    if (img_array)
    {
        stbi_image_free(img_array);
    }

    free(img_gray_array);
}

void Image::save_gray_img()
{
    stbi_write_jpg("codebar_test.jpg", width, height, nb_chan, img_array, 100);
}

void create_gray_array()
{

}

int Image::get_size()
{
    return width * height;
}


int main(void)
{
    Image image("codebar.jpg");

    image.save_gray_img();

    return 0;
}
