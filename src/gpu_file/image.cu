#include "hip/hip_runtime.h"
#include "image.hh"

#include <iostream>
#include <stdlib.h>

__global__ void compute_gray(unsigned char *d_gray_array,
        unsigned char *d_array, int width, int weight,
        int blockSize, int gridSize)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int img_size = width * weight;

    while (id < img_size)
    {
        d_gray_array[id] = d_array[id * 3] * 0.2989 +
            d_array[id * 3 + 1] * 0.5870 + d_array[id * 3 + 2] * 0.1140;

        id += blockSize * gridSize;
    }

}


__global__ void compute_sobel(unsigned char *d_sobel_x,
        unsigned char *d_sobel_y, unsigned char *d_gray_array, int width,
        int weight, int blockSize, int gridSize)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int img_size = width * weight;

    while (id < img_size)
    {
        // We will compute 9 pixel withe the next shape

        // 1 2 3
        // 4 5 6
        // 7 8 9

        // So we will compte only the pixel that are not in the border


        if (id <= width)
        {
            d_sobel_x[id] = 0;
            d_sobel_y[id] = 0;
        }
        else if (id % width == 0)
        {
            d_sobel_x[id] = 0;
            d_sobel_y[id] = 0;
        }
        else if (id % width == (width - 1))
        {
            d_sobel_x[id] = 0;
            d_sobel_y[id] = 0;
        }
        else if (id >= (img_size - width))
        {
            d_sobel_x[id] = 0;
            d_sobel_y[id] = 0;
        }
        else
        {
            int sum_x = 0;
            int sum_y = 0;

            sum_x -= d_gray_array[id - 1 - width];
            sum_x += d_gray_array[id + 1 - width];
            sum_x -= 2 * d_gray_array[id - 1];
            sum_x += 2 * d_gray_array[id + 1];
            sum_x -= d_gray_array[id - 1 + width];
            sum_x += d_gray_array[id + 1 + width];

            sum_y -= d_gray_array[id - 1 - width];
            sum_y -= 2 * d_gray_array[id - width];
            sum_y -= d_gray_array[id + 1 - width];
            sum_y += d_gray_array[id - 1 + width];
            sum_y += 2 * d_gray_array[id + width];
            sum_y += d_gray_array[id + 1 + width];

            sum_x = abs(sum_x);
            sum_y = abs(sum_y);

            d_sobel_x[id] = sum_x;
            d_sobel_y[id] = sum_y;

        }

        id += blockSize * gridSize;
    }
}



Image::Image(const char* path)
{

    img_array = stbi_load(path, &width, &height, &nb_chan, 0);

    if (img_array == NULL)
    {
        std::cout << "Error : can't open the image: " << path << "\n";
    }

    img_gray_array = new unsigned char[width * height];
    img_sobel_x_array = new unsigned char[width * height];
    img_sobel_y_array = new unsigned char[width * height];
}


Image::~Image()
{
    if (img_array)
    {
        stbi_image_free(img_array);
    }

    free(img_gray_array);
    free(img_sobel_x_array);
    free(img_sobel_y_array);
}


void Image::save_gray_img()
{
    stbi_write_jpg("../../img/codebar_gray.jpg", width, height, 1,
            img_gray_array, 100);
}


void Image::save_sobel_img()
{

    for (int i = 0; i < width * height; i++)
    {
        printf("x : %d y : %d\n", img_sobel_x_array[i], img_sobel_y_array[i]);
    }

    stbi_write_jpg("../../img/codebar_sobel_x.jpg", width, height, 1,
            img_sobel_x_array, 100);
    stbi_write_jpg("../../img/codebar_sobel_y.jpg", width, height, 1,
            img_sobel_y_array, 100);
}


void Image::create_gray_array()
{
    /*
    // CPU Version
    for (int i = 0; i < height * width; ++i)
    {
        img_gray_array[i] = (uint8_t)((img_array[i * 3] +
                    img_array[i * 3 + 1] + img_array[i * 3 + 2])/3.0);
    }
    */


    unsigned char *d_gray_img;
    unsigned char *d_img;

    size_t gray_img_size = width * height * sizeof(unsigned char);
    size_t img_size = width * height * 3 * sizeof(unsigned char);

    hipMalloc(&d_gray_img, gray_img_size);
    hipMalloc(&d_img, img_size);


    hipMemcpy( d_gray_img, img_gray_array, gray_img_size, hipMemcpyHostToDevice);
    hipMemcpy( d_img, img_array, img_size, hipMemcpyHostToDevice);

    int blockSize, gridSize;

    blockSize = 5;
    gridSize = 2;

    compute_gray<<<gridSize, blockSize>>>(d_gray_img, d_img, width, height,
                                            blockSize, gridSize);
    hipDeviceSynchronize();

    hipMemcpy(img_gray_array, d_gray_img, gray_img_size, hipMemcpyDeviceToHost);

}

void Image::create_sobel_array()
{
    unsigned char *d_sobel_x;
    unsigned char *d_sobel_y;
    unsigned char *d_gray_img;

    size_t img_size = width * height * sizeof(unsigned char);

    hipMalloc(&d_sobel_x, img_size);
    hipMalloc(&d_sobel_y, img_size);
    hipMalloc(&d_gray_img, img_size);


    hipMemcpy( d_sobel_x, img_sobel_x_array, img_size, hipMemcpyHostToDevice);
    hipMemcpy( d_sobel_y, img_sobel_y_array, img_size, hipMemcpyHostToDevice);
    hipMemcpy( d_gray_img, img_gray_array, img_size, hipMemcpyHostToDevice);

    int blockSize, gridSize;

    blockSize = 5;
    gridSize = 2;

    compute_sobel<<<gridSize, blockSize>>>(d_sobel_x, d_sobel_y, d_gray_img,
                                        width, height, blockSize, gridSize);
    hipDeviceSynchronize();

    hipMemcpy(img_sobel_x_array, d_sobel_x, img_size, hipMemcpyDeviceToHost);
    hipMemcpy(img_sobel_y_array, d_sobel_y, img_size, hipMemcpyDeviceToHost);

}



int Image::get_size()
{
    return width * height;
}


int main(void)
{
    Image image("../../img/codebar.jpg");
    image.create_gray_array();
    image.save_gray_img();
    image.create_sobel_array();
    image.save_sobel_img();

    /*
       int width, height, nb_chan;
       unsigned char *img = stbi_load("../../img/codebar.jpg", &width, &height, &nb_chan, 0);

       size_t img_size = width * height * nb_chan;
       printf("We have a size of %dx%d, %d\n", width, height, img_size);
       size_t gray_img_size = width * height;

       unsigned char *gray_img = (unsigned char *) malloc(gray_img_size);

       for (int i = 0; i < width * height; i++)
       {
       gray_img[i] = img[i * 3] * 0.2989 + img[i * 3 + 1] * 0.5870 + img[i * 3 + 2] * 0.1140;


       }

       stbi_write_jpg("../../img/img_gray_result.jpg", width, height, 1, gray_img, 100);
     */

    return 0;
}
